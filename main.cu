//
// Created by xiezheng on 2020/9/8.
//
#include <iostream>
#include <hip/hip_runtime.h>

#include <sys/time.h>
#include <math.h>
#define ROWS 1024
#define COLS 1024
//extern "C"
//{
//
//}
using namespace std;

__global__ void Plus(float A[], float B[], float C[],int n)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    C[i] = A[i] + B[i];

}

namespace test_cpu //在cpu计算
{
    void add_cpu_demo()
    {


        float *A,*B,*C;
        int n = 1024*1024;
        int size = n * sizeof(float);
        A = (float*)malloc(size);
        B = (float*)malloc(size);
        C = (float*)malloc(size);

        for (int i = 0; i < n; ++i) {
            A[i] = 90.0;
            B[i] = 10.0;
        }

        for (int j = 0; j < n; ++j) {
            C[j] = A[j] + B[j];
        }


        float max_error = 0.0;
        for (int k = 0; k < n; ++k) {
            max_error += fabs(100.0 - C[k]);
        }
        std::cout << "max_error is " << max_error << std::endl;

        delete A;
        delete B;
        delete C;

    }
}

namespace test_gpu
{
    void add_gpu_demo()
    {
        float *A, *B, *C, *Ad,*Bd,*Cd;
        int n = 1024*1024;
        int size = n* sizeof(int);
        A = (float*)malloc(n* sizeof(float));
        B = (float*)malloc(n* sizeof(float));
        C = (float*)malloc(n* sizeof(float));


        for (int i = 0; i < n; ++i) {
            A[i] = 90.0;
            B[i] = 10.0;
        }


        hipMalloc((void**)&Ad,size);
        hipMalloc((void**)&Bd,size);
        hipMalloc((void**)&Cd,size);

        hipMemcpy(Ad,A,size,hipMemcpyHostToDevice);
        hipMemcpy(Bd,B,size,hipMemcpyHostToDevice);
        hipMemcpy(Cd,C,size,hipMemcpyHostToDevice);

        dim3 dimBlock(512);
        dim3 dimGrid(n/512);

        Plus<<<dimGrid,dimBlock>>>(Ad,Bd,Cd,n);

        hipMemcpy(C,Cd,size,hipMemcpyHostToDevice);

        // 校验误差
        float max_error = 0.0;
        for(int i=0;i<n;i++)
        {
            max_error += fabs(100.0 - C[i]);
        }
        cout << "max error is " << max_error << endl;

        hipFree(Ad);
        hipFree(Bd);
        hipFree(Cd);

        delete A;
        delete B;
        delete C;


    }
}


int main()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    std::cout<<"devices count = "<<deviceCount<<std::endl;

    for (int i = 0; i < deviceCount; ++i) {
        hipDeviceProp_t devProp;
        hipGetDeviceProperties(&devProp,i);
        std::cout << "使用GPU device " << i << ": " << devProp.name << std::endl;
        std::cout << "设备全局内存总量： " << devProp.totalGlobalMem / 1024 / 1024 << "MB" << std::endl;
        std::cout << "SM的数量：" << devProp.multiProcessorCount << std::endl;
        std::cout << "每个线程块的共享内存大小：" << devProp.sharedMemPerBlock / 1024.0 << " KB" << std::endl;
        std::cout << "每个线程块的最大线程数：" << devProp.maxThreadsPerBlock << std::endl;
        std::cout << "设备上一个线程块（Block）种可用的32位寄存器数量： " << devProp.regsPerBlock << std::endl;
        std::cout << "每个EM的最大线程数：" << devProp.maxThreadsPerMultiProcessor << std::endl;
        std::cout << "每个EM的最大线程束数：" << devProp.maxThreadsPerMultiProcessor / 32 << std::endl;
        std::cout << "设备上多处理器的数量： " << devProp.multiProcessorCount << std::endl;
        std::cout << "======================================================" << std::endl;

    }

    struct timeval start,end;
    gettimeofday(&start,NULL);

//    test_cpu::add_cpu_demo();
    test_gpu::add_gpu_demo();
    gettimeofday( &end, NULL );
    int timeuse = 1000000 * ( end.tv_sec - start.tv_sec ) + end.tv_usec - start.tv_usec;
    std::cout << "total time is " << timeuse/1000 << "ms" <<std::endl;


    return 0;

}